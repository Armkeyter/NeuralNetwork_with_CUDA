#include "hip/hip_runtime.h"
#include <iostream>

#include "./cuda_kernel.cuh"



void getDeviceInfo() {
    const int kb = 1024;
    const int mb = kb * kb;
    int device;
    hipGetDeviceCount(&device);
    hipDeviceProp_t props;
    if (device < 1) {
        std::cout << "No GPU was found" << std::endl;
    }
    else {
        hipGetDeviceProperties(&props, 0);
        std::cout << props.name << ": " << props.major << "." << props.minor << std::endl;
        std::cout << "Warp Size: \t\t" << props.warpSize << std::endl;
        std::cout << "Threads Per Block: \t" << props.maxThreadsPerBlock << std::endl;
        std::cout << "Max Block dim: \t\t" << props.maxThreadsDim[0] << ',' << props.maxThreadsDim[1] << ',' << props.maxThreadsDim[2] << std::endl;
        std::cout << "Max Grid dim: \t\t" << props.maxGridSize[0] << ',' << props.maxGridSize[1] << ',' << props.maxGridSize[2] << std::endl;
        std::cout << "Global Memory: \t\t" << props.totalGlobalMem / mb << "mb" << std::endl;
        std::cout << "Shared Memory: \t\t" << props.sharedMemPerBlock / kb << "kb" << std::endl;
        std::cout << "Constant Memory: \t" << props.totalConstMem / kb << "kb" << std::endl;
        std::cout << "Block Registers: \t" << props.regsPerBlock << std::endl << std::endl;
    }
}


__device__ void dev_matmul(float* A, float* B, float* C, int rowsA, int colsA, int colsB) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((Row < rowsA) && (Col < colsB)) {
        float Cvalue = 0;
        for (int k = 0; k < colsA; ++k)
            Cvalue += A[Row * colsA + k] * B[k * colsB + Col];
        C[Row * colsB + Col] = Cvalue;
    }
}

__device__ void addbias(float* data, float* biases,float* result, int rows, int cols) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((Row < rows) && (Col < cols)) {
        result[Row * cols + Col] = data[Row * cols + Col] + biases[Col];
    }
    
}


__global__ void matmul(float* A, float* B, float* C, int rowsA, int colsA, int colsB) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((Row < rowsA) && (Col < colsB)) {
        float Cvalue = 0;
        for (int k = 0; k < colsA; ++k)
            Cvalue += A[Row * colsA + k] * B[k * colsB + Col];
        C[Row * colsB + Col] = Cvalue;
    }
}


__global__ void forwardProp(float* A, float* B,float* biases, float* C, int rowsA, int colsA, int colsB) {
    dev_matmul(A, B, C, rowsA, colsA, colsB);
    addbias(C, biases, C, rowsA, colsB);
}

__global__ void sigmoidCU(float* A, float* B,bool is_derivative, int rows, int cols) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((Row < rows) && (Col < cols)) {
        if (!is_derivative)
            B[Row * cols + Col] = 1 / (1 + exp(-A[Row * cols + Col]));
        else
            B[Row * cols + Col] = A[Row * cols + Col] * (1 - A[Row * cols + Col]);
    }
}

__global__ void softamxCU(float* A, float* B, int rows, int cols) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    if ((Row < rows) && (Col < cols)) {
        B[Row * cols + Col] = 1 / (1 + exp(-A[Row * cols + Col]));
    }
}

void matrixMultiplication(int threadsN, float* data_GPU, float* weights_GPU, float* result,
    int rowsX, int colsX, int rowsWeights) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block

    dim3 threadsPerBlock(threadsN, threadsN);
    dim3 blocksPerGrid((rowsWeights - 1) / threadsN + 1, (rowsX - 1) / threadsN + 1, 1);
    if (threadsN * threadsN > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(threadsN) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(threadsN) / double(threadsPerBlock.y));
    }
    matmul<<<blocksPerGrid, threadsPerBlock>>>(data_GPU, weights_GPU, result, rowsX, colsX, rowsWeights);
}

void sigmoid(int threadsN, float* data_GPU, float* reuslts_GPU, int rows, int cols, bool is_derivative)
{
    dim3 threadsPerBlock(threadsN, threadsN);
    dim3 blocksPerGrid((rows - 1) / threadsN + 1, (rows - 1) / threadsN + 1, 1);
    if (threadsN * threadsN > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(threadsN) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(threadsN) / double(threadsPerBlock.y));
    }
    sigmoidCU<<<blocksPerGrid, threadsPerBlock>>>(data_GPU, reuslts_GPU, is_derivative, rows, cols);
}

void softmax(int threadsN, float* data_GPU, float* reuslts_GPU, int rows, int cols)
{
    dim3 threadsPerBlock(threadsN, threadsN);
    dim3 blocksPerGrid((rows - 1) / threadsN + 1, (rows - 1) / threadsN + 1, 1);
    if (threadsN * threadsN > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(threadsN) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(threadsN) / double(threadsPerBlock.y));
    }
    softamxCU<<<blocksPerGrid, threadsPerBlock>>>(data_GPU, reuslts_GPU, rows, cols);
}


void forwardPropagation(int threadsN, float* data_GPU, float* weights_GPU,float *biases, float* result,
    int rowsX, int colsX, int rowsWeights) {

    // declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block

    dim3 threadsPerBlock(threadsN, threadsN);
    dim3 blocksPerGrid((rowsWeights - 1) / threadsN + 1, (rowsX - 1) / threadsN + 1, 1);
    if (threadsN * threadsN > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(threadsN) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(threadsN) / double(threadsPerBlock.y));
    }

    forwardProp<<<blocksPerGrid, threadsPerBlock>>>(data_GPU, weights_GPU, biases, result, rowsX, colsX, rowsWeights);
}
